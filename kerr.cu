
#include "vector_functions.h"
#include <cmath>
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#ifndef M_C
#define M_C 299792458.0f
#endif

// ---------------------------------------------------------------------
// Device constants (set at compile time; you may also update via
// cudaMemcpyToSymbol)
// Device constants (set at compile time; you may also update via
// cudaMemcpyToSymbol)
__constant__ float c_a = 0.6f;
__constant__ float c_rs = 1;
__constant__ unsigned int c_num_steps = 15000;
__constant__ unsigned int c_layers = 1;
__constant__ float c_M = 1.0f;        // Mass parameter
__constant__ float c_epsilon = 1e-10; // Numerical tolerance
__constant__ float3 worldUp = {0.0f, 1.0f, 0.0f};

// Additional simulation parameters
__constant__ float c_h = 0.1f;           // Integration step size
__constant__ double r_env_val = 1000.0f; // Environment boundary

// ---------------------------------------------------------------------
// Coordinate convertion functions

// helper math (as before)
__device__ float3 crossf3(const float3 &a, const float3 &b) {
  return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
                     a.x * b.y - a.y * b.x);
}

__device__ float3 normalizef3(const float3 &v) {
  float len2 = v.x * v.x + v.y * v.y + v.z * v.z;
  float invLen = 1.0f / sqrtf(len2);
  return make_float3(v.x * invLen, v.y * invLen, v.z * invLen);
}

__device__ inline float3 operator*(const float3 &v, float s) {
  return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ inline float3 operator*(float s, const float3 &v) {
  return make_float3(v.x * s, v.y * s, v.z * s);
}

__device__ inline float3 operator+(const float3 &a, const float3 &b) {
  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 spherical_to_cartesian(float r, float theta, float phi) {
  return make_float3(r * sinf(theta) * cosf(phi), r * sinf(theta) * sinf(phi),
                     r * cosf(theta));
}

__device__ void cartesian_to_boyer_lindquist(float x, float x_vel, float y,
                                             float y_vel, float z, float z_vel,
                                             float A, float *out) {
  double r2 = x * x + y * y + z * z;
  double A2 = A * A;
  double root = sqrt(A2 * (A2 - 2.0 * (x * x + y * y) + 2.0 * z * z) + r2 * r2);
  double radius = sqrt((-A2 + r2 + root) * 0.5);

  float azimuthal_angle = atan2f(y, x);
  float polar_angle = acosf(z / radius);

  double denom = 2.0 * radius * radius + A2 - r2;
  double radius_velocity =
      ((radius * (x * x_vel + y * y_vel + z * z_vel)) / denom +
       A2 * z * z_vel / (radius * denom));

  float polar_denom = radius * sqrtf(radius * radius - z * z);
  float polar_velocity = (z * radius_velocity - z_vel * radius) / polar_denom;

  float azimuthal_velocity = (y_vel * x - x_vel * y) / (x * x + y * y);

  out[0] = radius;
  out[1] = radius_velocity;
  out[2] = polar_angle;
  out[3] = polar_velocity;
  out[4] = azimuthal_angle;
  out[5] = azimuthal_velocity;
}

// ---------------------------------------------------------------------
// Kerr metric helper functions

__device__ float sigma(float r, float theta) {
  float cos_theta = cos(theta);
  return r * r + c_a * c_a * cos_theta * cos_theta;
}

__device__ float delta_r(float r) { return r * r + c_a * c_a - 2.0f * c_M * r; }

__device__ float ddelta_r(float r) { return 2.0f * (r - c_M); }

// ---------------------------------------------------------------------
// Functions W_r, W_theta and their derivatives

__device__ float W_r(float r, float E, float L) {
  return E * (r * r + c_a * c_a) - c_a * L;
}

__device__ float dWsquare_r(float r, float E, float L) {
  float W = W_r(r, E, L);
  float dW_dr = 2.0f * E * r;
  return 2.0f * W * dW_dr;
}

__device__ float W_theta(float theta, float E, float L) {
  float sin_theta = sin(theta);
  sin_theta = fmax(sin_theta, c_epsilon);
  return c_a * E * sin_theta - L / sin_theta;
}

__device__ float dWsquare_theta(float theta, float E, float L) {
  float sin_theta = sin(theta);
  float cos_theta = cos(theta);
  sin_theta = fmax(sin_theta, c_epsilon);
  float dW_dtheta = cos_theta * (c_a * E + L / (sin_theta * sin_theta));
  return 2.0f * W_theta(theta, E, L) * dW_dtheta;
}

// ---------------------------------------------------------------------
// Definitions of the conserved quantities and derived functions

__device__ float E_func(float r, float theta, float dr, float dtheta,
                        float dphi) {
  float sin_theta = sin(theta);
  sin_theta = fmax(sin_theta, c_epsilon);
  float delta = delta_r(r);
  float term = ((c_a * c_a * sin_theta * sin_theta - delta) *
                    (-dr * dr / delta - dtheta * dtheta) +
                (dphi * sin_theta) * (dphi * sin_theta) * delta);
  return sqrt(term);
}

__device__ float L_func(float r, float theta, float dphi, float E) {
  float sin_theta = sin(theta);
  sin_theta = fmax(sin_theta, c_epsilon);
  float delta = delta_r(r);
  float sigma_val = sigma(r, theta);
  float num = c_a * E * delta +
              (sigma_val * delta * dphi - c_a * E * (r * r + c_a * c_a));
  float denom = delta - c_a * c_a * sin_theta * sin_theta;
  return sin_theta * sin_theta * num / denom;
}

__device__ float k_func(float r, float theta, float dr, float E, float L) {
  float sigma_val = sigma(r, theta);
  float delta = delta_r(r);
  float W = W_r(r, E, L);
  return (W * W - sigma_val * sigma_val * dr * dr) / delta;
}

// ---------------------------------------------------------------------
// Geodesic equations: state vector y = [r, theta, phi, p_r, p_theta]
__device__ float dr_func(float r, float theta, float p_r) {
  return delta_r(r) * p_r / sigma(r, theta);
}

__device__ float dtheta_func(float r, float theta, float p_theta) {
  return p_theta / sigma(r, theta);
}

__device__ float dphi_func(float r, float theta, float E, float L) {
  float sig = sigma(r, theta);
  float delta = delta_r(r);
  float sin_theta = sin(theta);
  sin_theta = fmax(sin_theta, c_epsilon);
  return (c_a * W_r(r, E, L) / delta - W_theta(theta, E, L) / sin_theta) / sig;
}

__device__ float dp_r(float r, float theta, float p_r, float E, float L,
                      float k_val) {
  float sig = sigma(r, theta);
  float delta = delta_r(r);
  float d_delta = ddelta_r(r);
  float dW2 = dWsquare_r(r, E, L);
  float num = dW2 - d_delta * k_val;
  return (num / (2.0f * delta) - d_delta * p_r * p_r) / sig;
}

__device__ float dp_theta(float r, float theta, float E, float L) {
  float sig = sigma(r, theta);
  float dW_theta_val = dWsquare_theta(theta, E, L);
  return -dW_theta_val / (2.0f * sig);
}

__device__ float Ke(float k, float E) { return k / (E * E); }

__device__ float aLE(float L, float E) { return c_a * L / E; }

// ---------------------------------------------------------------------
// RK4 integration using c_a loop to compute k coefficients
// The state vector y has 5 components.
__device__ void rk4(float *y, float h, float E, float L, float k_val) {
  float k[4][5];   // k coefficients for the 4 stages
  float y_temp[5]; // temporary storage

  // Loop over the 4 stages
#pragma unroll
  for (int stage = 0; stage < 4; ++stage) {
    float factor = (stage == 0) ? 0.0f : (stage == 3 ? 1.0f : 0.5f);
    // Compute temporary state: y_temp = y + factor * h * (previous k)
    // For stage 0 we simply have y_temp = y.
#pragma unroll
    for (int i = 0; i < 5; ++i)
      y_temp[i] = y[i] + (stage == 0 ? 0.0f : factor * h * k[stage - 1][i]);

    // Compute the derivatives at y_temp
    k[stage][0] = dr_func(y_temp[0], y_temp[1], y_temp[3]);
    k[stage][1] = dtheta_func(y_temp[0], y_temp[1], y_temp[4]);
    k[stage][2] = dphi_func(y_temp[0], y_temp[1], E, L);
    k[stage][3] = dp_r(y_temp[0], y_temp[1], y_temp[3], E, L, k_val);
    k[stage][4] = dp_theta(y_temp[0], y_temp[1], E, L);
  }
  // Combine the stages
#pragma unroll
  for (int i = 0; i < 5; ++i) {
    y[i] += h / 6.0f * (k[0][i] + 2.0f * k[1][i] + 2.0f * k[2][i] + k[3][i]);
  }
}

// ---------------------------------------------------------------------
// Kernel: each thread simulates one ray.
// Input initial conditions are in the order:
// [r0, theta0, phi0, dr0, dtheta0, dphi0]
// The output trajectory (state vector per step) and the number of steps per ray
// are stored in contiguous device memory.
__global__ void simulateRayKernel(float3 pos, size_t num_rays_per_dim,
                                  float *trajectories, int *steps_out) {
  // printf("%.2f %.2f %.2f \n", pos.x ,pos.y, pos.z);
  int const idx = blockIdx.x * blockDim.x + threadIdx.x;
  int const num_rays = num_rays_per_dim * num_rays_per_dim;

  if (idx >= num_rays)
    return;

  int const idx_theta = idx / num_rays_per_dim;
  int const idx_phi = idx % num_rays_per_dim;

  float theta = (M_PI * idx_theta) / num_rays_per_dim;
  float phi = (2.0f * M_PI);
  // @TODO: (Investigate); Might need to rotate outgoing dirs to account for
  // camera orientation
  float3 camPos = make_float3(pos.x, pos.y, pos.z); // camera world pos
  float3 forward =
      normalizef3(make_float3(-camPos.x, // since modelCenter == (0,0,0)
                              -camPos.y, -camPos.z));

  float3 right = normalizef3(crossf3(forward, worldUp));
  float3 upVec = crossf3(right, forward);

  // now build your ray as before:
  float sinT = sinf(theta), cosT = cosf(theta);
  float sinP = sinf(phi), cosP = cosf(phi);

  float3 dir = sinT * (cosP * right + sinP * upVec) + cosT * forward;

  dir = normalizef3(dir);
  //   printf("%.2f %.2f %.2f \n", dir.x, dir.y, dir.z);

  float const x_vel = M_C * dir.x;
  float const y_vel = M_C * dir.y;
  float const z_vel = M_C * dir.z;

  float const A = c_a * c_rs / 2;

  float bl[6];
  cartesian_to_boyer_lindquist(pos.x, x_vel, pos.y, y_vel, pos.z, z_vel, A, bl);

  float const r0 = 2.0f / c_rs * bl[0];
  float const theta0 = bl[2];
  float const phi0 = bl[4];
  float const dr0 = bl[1] / M_C;
  float const dtheta0 = bl[3] * c_rs / (2.0f * M_C);
  float const dphi0 = bl[5] * c_rs / (2.0f * M_C);

  printf("%.2f %.2f %.2f %.2f %.2f %.2f \n", r0, theta0, phi0, dr0, dtheta0,
         dphi0);

  // Compute conserved quantities using Kerr equations.
  float E = E_func(r0, theta0, dr0, dtheta0, dphi0);
  float L = L_func(r0, theta0, dphi0, E);
  float k_val = k_func(r0, theta0, dr0, E, L);

  // Compute initial momenta.
  float S = sigma(r0, theta0);
  float p_r0 = S * dr0 / delta_r(r0);
  float p_theta0 = S * dtheta0;

  // Set up the initial state vector: [r, theta, phi, p_r, p_theta]
  float y[5];
  y[0] = r0;
  y[1] = theta0;
  y[2] = phi0;
  y[3] = p_r0;
  y[4] = p_theta0;
  // printf("%.2f, %.2f\n", theta0, phi0);

  // Pointer to this ray's trajectory data.
  float *ray_traj = &trajectories[idx * c_num_steps * 5];

  int step;
  for (step = 0; step < c_num_steps; step++) {
    // Terminate integration if ray is inside the horizon or outside the
    // environment.
    if (y[0] < 1.99 || y[0] > r_env_val)
      break;

    // Store the current state into the trajectory.
    for (int j = 0; j < 5; j++) {
      ray_traj[step * 5 + j] = y[j];
    }
    // Advance one RK4 step.
    rk4(y, c_h, E, L, k_val);
  }
  steps_out[idx] = step;
}

// ---------------------------------------------------------------------
// Exported function for DLL interface
// This function is called from Python via a DLL (or shared library).
// It accepts the number of rays, number of integration steps, and an array
// of initial conditions (size: num_rays * 6). It outputs the trajectory data
// (num_rays * num_steps * 5 double values) and the number of steps for each
// ray.
extern "C" {

#ifdef _WIN32
__declspec(dllexport)
#endif
void simulateRays(float x, float y, float z, const float rs, const float Kerr,
    unsigned int num_rays_per_dim, unsigned int num_steps,
    float *trajectories_host, int *steps_out_host) {

  size_t num_rays = num_rays_per_dim * num_rays_per_dim;
  // Calculate sizes for memory allocation.
  size_t traj_size = num_rays * num_steps * 5 * sizeof(float);
  size_t steps_size = num_rays * sizeof(int);

  hipMemcpyToSymbol(HIP_SYMBOL(c_a), &Kerr, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(c_rs), &rs, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(c_num_steps), &num_steps, sizeof(unsigned int));

  // Allocate device memory.
  float *d_trajectories = nullptr;
  int *d_steps_out = nullptr;
  hipMalloc(&d_trajectories, traj_size);
  hipMalloc(&d_steps_out, steps_size);

  // Determine kernel launch configuration.
  int threadsPerBlock = 256;
  int blocks = (int)((num_rays + threadsPerBlock - 1) / threadsPerBlock);

  // Launch the simulation kernel.
  simulateRayKernel<<<blocks, threadsPerBlock>>>(
      make_float3(x, y, z), num_rays_per_dim, d_trajectories, d_steps_out);
  hipDeviceSynchronize();

  // Copy the results back to host.
  hipMemcpy(trajectories_host, d_trajectories, traj_size,
             hipMemcpyDeviceToHost);
  hipMemcpy(steps_out_host, d_steps_out, steps_size, hipMemcpyDeviceToHost);

  // Free device memory.
  hipFree(d_trajectories);
  hipFree(d_steps_out);
}

} // extern "C"