#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include ""

__constant__ double rs = 1;

__device__ void geodesic(double u, double du, double& out_u, double& out_du) {
    double temp_u = u;
    out_u = du;
    out_du = -temp_u * (1 - (3 / 2) * rs * temp_u);
}

__device__ void rk4_step(double& u, double& du, double& phi, double h) {
    double k1_u, k1_du, k2_u, k2_du, k3_u, k3_du, k4_u, k4_du;

    geodesic(u, du, k1_u, k1_du);
    geodesic(u + 0.5 * k1_u * h, du + 0.5 * k1_du * h, k2_u, k2_du);
    geodesic(u + 0.5 * k2_u * h, du + 0.5 * k2_du * h, k3_u, k3_du);
    geodesic(u + k3_u * h, du + k3_du * h, k4_u, k4_du);

    phi += h;
    u = u + (k1_u + 2 * k2_u + 2 * k3_u + k4_u) * h / 6;
    du = du + (k1_du + 2 * k2_du + 2 * k3_du + k4_du) * h / 6;
}

__global__ void solve_geodesic_kernel(double u_0, double* du_0_values, double h, int num_paths, int num_steps, double* u_values, double* du_values, double* phi_values, double* angles_out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    double u = u_0;
    double du = du_0_values[idx];
    double phi = 0.0;

    // Perform the first RK4 step
    rk4_step(u, du, phi, h);

    double r = 1.0 / u;
    double r_0 = 1.0 / u_0;
    double a = r * sin(phi);
    double b = r * cos(phi) - r_0;

    // Store starting angle
    angles_out[idx * 3] = atan2(a, b);

    u_values[idx * num_steps] = u;
    du_values[idx * num_steps] = du;
    phi_values[idx * num_steps ] = phi;

    for (int step = 1; step < num_steps; step++) {
        
        if (1 / u < rs) break;
        if (1 / u > 30.0) break;
        rk4_step(u, du, phi, h);
        if (1 / u > 30.0) break;
        if (1 / u < rs) break;
        
        u_values[idx * num_steps + step] = u;
        du_values[idx * num_steps + step] = du;
        phi_values[idx * num_steps + step] = phi;
    }
    angles_out[idx * 3 + 1] = phi;
    angles_out[idx * 3 + 2] = u;
    printf("r = %f \n", 1/u);
}

extern "C" {
    __declspec(dllexport) void cuda_test(
        int num_paths, int num_steps, double u_0,
        double* du_0_values, double h, double* u_out, double* phi_out, double* angle_out) {

        // Allocate device memory
        double* d_du_0_values;
        double* d_u_values;
        double* d_du_values;
        double* d_phi_values;
        double* d_angle_values;

        hipMalloc(&d_du_0_values, num_paths * sizeof(double));
        hipMalloc(&d_u_values, num_paths * num_steps * sizeof(double));
        hipMalloc(&d_du_values, num_paths * num_steps * sizeof(double));
        hipMalloc(&d_phi_values, num_paths * num_steps * sizeof(double));
        hipMalloc(&d_angle_values, num_paths * 3 * sizeof(double));


        // Copy initial velocity values to device
        hipMemcpy(d_du_0_values, du_0_values, num_paths * sizeof(double), hipMemcpyHostToDevice);

        // Launch kernel
        int threadsPerBlock = 256;
        int numBlocks = (num_paths + threadsPerBlock - 1) / threadsPerBlock;
        solve_geodesic_kernel << <numBlocks, threadsPerBlock >> > (u_0, d_du_0_values, h, num_paths, num_steps, d_u_values, d_du_values, d_phi_values, d_angle_values);

        // Copy results back to host
        hipMemcpy(u_out, d_u_values, num_paths * num_steps * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(phi_out, d_phi_values, num_paths * num_steps * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(angle_out, d_angle_values, num_paths * 3 * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_du_0_values);
        hipFree(d_u_values);
        hipFree(d_du_values);
        hipFree(d_phi_values);
        hipFree(d_angle_values);
    }
}
